//(c) 2017 John Freeman and Jose Rivas

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define INITIAL_CAPACITY 1024

/******************** Find the min value **************************/
__global__ void minCompare(int *a, int *set, bool *check, int *capacity) {
    int cap = capacity[0];
    int offset = set[0];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int tabx = idx + cap + offset;
    int taby = idy + cap + offset;

    if (idx == idy) { return; }

    int xval = a[tabx];
    int yval = a[taby];
    
    if(yval == 0) {}
    else if (xval == 0) {
        check[idx] = false;
    } else if (xval > yval) {
        check[idx] = false;
    }
}

__global__ void cudaMin(int *a, int *set, bool *check, int* min, int *capacity) {
    int idx = blockIdx.x;

    if (check[idx]) {
        min[0] = a[idx + capacity[0] + set[0]];
    }
}

/************************* Find the max value **********************/
__global__ void maxCompare(int *a, bool *check) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx == idy) { return; }

    int xval = a[idx];
    int yval = a[idy];

    if (xval < yval) {
        check[idx] = false;
    }
}

__global__ void cudaMax(int *a, bool *check, int* max) {
    int idx = blockIdx.x;

    if (check[idx]) {
        max[0] = a[idx];
    }
}

/*********************** Helper Methods ********************************************/
__global__ void cudaBoolFill(bool *arr, int length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < length) {
        arr[i] = true;
    }
}

/********************** Min and Max Functions ******************************************/
void findMin(int *arr, const int length, const int offset, int *minimum, int *capacity) {
    //length - 1 = row, offset = location of first element

    bool *check;
    int *set;
    int *row = (int*) malloc(sizeof(int));
    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&set, intSize);
    hipMemcpy(set, (int*)&offset, intSize, hipMemcpyHostToDevice);

    hipMemcpy(row, capacity, intSize, hipMemcpyDeviceToHost);
    row[0] = row[0] * (length - 1);
   
    printf("offset = %d    length = %d     row = %d\n", offset, length, row[0]);

    int *row2;
    hipMalloc((void**) &row2, intSize);
    hipMemcpy(row2, row, intSize, hipMemcpyHostToDevice);

    minCompare<<< dim3(length, length), 1 >>>(arr, set, check, row2);
    cudaMin<<< dim3(length, 1), 1 >>>(arr, set, check, minimum, row2);

    hipFree(check);
}

int findMax(int *arr, const int length) {
    bool *check;
    int *max;

    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&max, intSize);

    maxCompare<<< dim3(length, length), 1 >>>(arr, check);
    cudaMax<<< dim3(length, 1), 1 >>>(arr, check, max);

    int maxhost[1];
    hipMemcpy(maxhost, max, intSize, hipMemcpyDeviceToHost);

    hipFree(max);
    hipFree(check);

    return maxhost[0];
}

/********************* Find the Curl *****************************************/
int findCurl(int *sequence, int *table, int length, int capacity){
    int *tempResults;
    hipMalloc((void **) &tempResults, (length >> 1) * sizeof(int));
    int *cap;
    hipMalloc((void **) &cap, sizeof(int));
    hipMemcpy(cap, (int*)&capacity, sizeof(int), hipMemcpyHostToDevice);

    for(int i(0); i < (length >> 1); ++i) {
        //int *p = &(table[i][(length - 1) - i]);
        //findMin(p, length, &(tempResults[i]));
        findMin(table, i+1, (length - 1) - i, &(tempResults[i]), cap);
    }

    int *results = (int *) malloc((length >> 1) * sizeof(int));
    hipMemcpy(results, tempResults, (length >> 1) * sizeof(int), hipMemcpyDeviceToHost);
    for(int i(0); i < (length >> 1); ++i) {
        printf("%d ", results[i]);
    }
    printf("\n");

    int curl = findMax(tempResults, length);

    hipFree(tempResults);

    return curl;
}

void printTable(int *table, int length, int capacity) {
    int *CPUTable;
    CPUTable = (int *) malloc(capacity * capacity * sizeof(int));
    hipMemcpy(CPUTable, table, capacity * capacity * sizeof(int), hipMemcpyDeviceToHost);

    for(int i(0); i < length; ++i) {
        for(int j(0); j < length; ++j) {
            printf("%d ", CPUTable[(i * capacity) + j]);
        }
        printf("\n");
    }

    free(CPUTable);
}

__global__ void fillColumn(int *sequence, int *table, int *seqPosition, int *cap) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int index = *seqPosition;
    int capacity = *cap;
    int value = 1;
    
    if(row == index){}
    else if(sequence[index - (row + 1)] == sequence[index]) {
        int t = table[(row * capacity) + (index - (row + 1))];
        if(t == 0) {
            value = 2;
        } else {
            value = table[(row * capacity) + (index - (row + 1))] + 1;
        }
    }

    table[(row * capacity) + index] = value;
}

void initializeTable(int *sequence, int *table, int length, int capacity) {

    int *index;
    hipMalloc((void **)&index, sizeof(int));
    int *cap;
    hipMalloc((void **)&cap, sizeof(int));
    hipMemcpy(cap, (void *)&capacity, sizeof(int), hipMemcpyHostToDevice);

    for(int i(0); i < length; ++i) {
        hipMemcpy(index, (void *)&i, sizeof(int), hipMemcpyHostToDevice);
        fillColumn<<< dim3(i + 1, 1), 1 >>>(sequence, table, index, cap);
    }

    hipFree(index);
}

int main() {
    int *table;
    int capacity = INITIAL_CAPACITY;

    hipMalloc((void**)&table, (INITIAL_CAPACITY * INITIAL_CAPACITY) * sizeof(int));

    while (1) {

        hipMemset(table, 0, (capacity * capacity) * sizeof(int));
        
        char buffer[100];
        printf("Input a sequence to curl:\n");
        scanf("%s", buffer);

        int i(0);
        int sequence[INITIAL_CAPACITY];
        for (; buffer[i] != '\0'; ++i) {
            sequence[i] = buffer[i] - '0';
        }

        int seqLength = i;
        int sequenceByteSize = seqLength * sizeof(int);
        int *cudaSequence;
        hipMalloc((void**)&cudaSequence, sequenceByteSize);
        hipMemcpy(cudaSequence, sequence, sequenceByteSize, hipMemcpyHostToDevice);

        initializeTable(cudaSequence, table, seqLength, capacity);

        clock_t start = clock();

        int *size;
        hipMalloc((void**)&size, sizeof(int));
        int *cap;
        hipMalloc((void **)&cap, sizeof(int));
        hipMemcpy(cap, (void *)&capacity, sizeof(int), hipMemcpyHostToDevice);
        int curl = (seqLength == 1) ? 1: 0;

        while(curl != 1) {
            curl = findCurl(cudaSequence, table, seqLength, capacity);
            printf("curl = %d\n", curl);
            printTable(table, seqLength, capacity);
            sequence[seqLength] = curl;
            hipMemcpy(size, (int*)&seqLength, sizeof(int), hipMemcpyHostToDevice);
            sequenceByteSize = ++seqLength * sizeof(int);
            hipMalloc((void**)&cudaSequence, sequenceByteSize);
            hipMemcpy(cudaSequence, sequence, sequenceByteSize, hipMemcpyHostToDevice);
            fillColumn<<< dim3(seqLength, 1), 1 >>>(cudaSequence, table, size, cap);
        }

        clock_t stop = clock();
        double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
        printf("Elapsed time: %.3fs\n", elapsed);
        printf("curl is %d\n\nsequence = ", curl);

        for(i = 0; i < seqLength; ++i){
            printf("%d ", sequence[i]);
        }

        printf("\n\n");
        hipFree(cudaSequence);
    }
    return 0;
}
