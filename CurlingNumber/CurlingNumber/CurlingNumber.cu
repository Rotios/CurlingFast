#include "hip/hip_runtime.h"
//(c) 2017 John Freeman and Jose Rivas

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>

#define INITIAL_CAPACITY 1024
#define X(r, c) ((r * r) + r) * 0.5 + c 

/******************** Find the min value **************************/
__global__ void minCompare(int *a, int *set, bool *check, int *capacity) {
    int cap = capacity[0];
    int offset = set[0];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int tabx = idx + cap + offset;
    int taby = idy + cap + offset;

    if (idx == idy) { return; }

    int xval = a[tabx];
    int yval = a[taby];
    
    if(yval == 0) {}
    else if (xval == 0) {
        check[idx] = false;
    } else if (xval > yval) {
        check[idx] = false;
    }
}

__global__ void cudaMin(int *a, int *set, bool *check, int* min, int *capacity) {
    int idx = blockIdx.x;

    if (check[idx]) {
        min[0] = a[idx + capacity[0] + set[0]];
    }
}

/************************* Find the max value **********************/
__global__ void maxCompare(int *a, bool *check) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx == idy) { return; }

    int xval = a[idx];
    int yval = a[idy];

    if (xval < yval) {
        check[idx] = false;
    }
}

__global__ void cudaMax(int *a, bool *check, int* max) {
    int idx = blockIdx.x;

    if (check[idx]) {
        max[0] = a[idx];
    }
}

/*********************** Helper Methods ********************************************/
__global__ void cudaBoolFill(bool *arr, int length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < length) {
        arr[i] = true;
    }
}

/********************** Min and Max Functions ******************************************/
void findMin(int *arr, const int length, const int offset, int *minimum, int *capacity) {
    //length - 1 = row, offset = location of first element

    bool *check;
    int *set;
    int *row = (int*) malloc(sizeof(int));
    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&set, intSize);
    hipMemcpy(set, (int*)&offset, intSize, hipMemcpyHostToDevice);

    hipMemcpy(row, capacity, intSize, hipMemcpyDeviceToHost);
    row[0] = row[0] * (length - 1);
   
    printf("offset = %d    length = %d     row = %d\n", offset, length, row[0]);

    int *row2;
    hipMalloc((void**) &row2, intSize);
    hipMemcpy(row2, row, intSize, hipMemcpyHostToDevice);

    minCompare<<< dim3(length, length), 1 >>>(arr, set, check, row2);
    cudaMin<<< dim3(length, 1), 1 >>>(arr, set, check, minimum, row2);

    hipFree(check);
}

int findMax(int *arr, const int length) {
    bool *check;
    int *max;

    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&max, intSize);

    maxCompare<<< dim3(length, length), 1 >>>(arr, check);
    cudaMax<<< dim3(length, 1), 1 >>>(arr, check, max);

    int maxhost[1];
    hipMemcpy(maxhost, max, intSize, hipMemcpyDeviceToHost);

    hipFree(max);
    hipFree(check);

    return maxhost[0];
}

/********************* Find the Curl *****************************************/
int findCurl(int *sequence, int *table, int length, int capacity){
    int *tempResults;
    hipMalloc((void **) &tempResults, (length >> 1) * sizeof(int));
    int *cap;
    hipMalloc((void **) &cap, sizeof(int));
    hipMemcpy(cap, (int*)&capacity, sizeof(int), hipMemcpyHostToDevice);

    for(int i(0); i < (length >> 1); ++i) {
        //int *p = &(table[i][(length - 1) - i]);
        //findMin(p, length, &(tempResults[i]));
        findMin(table, i+1, (length - 1) - i, &(tempResults[i]), cap);
    }

    int *results = (int *) malloc((length >> 1) * sizeof(int));
    hipMemcpy(results, tempResults, (length >> 1) * sizeof(int), hipMemcpyDeviceToHost);
    for(int i(0); i < (length >> 1); ++i) {
        printf("%d ", results[i]);
    }
    printf("\n");

    int curl = findMax(tempResults, length);

    hipFree(tempResults);

    return curl;
}

void printTable(int *table, int length, int capacity) {
    int *CPUTable;
    CPUTable = (int *) malloc(capacity * capacity * sizeof(int));
    hipMemcpy(CPUTable, table, capacity * capacity * sizeof(int), hipMemcpyDeviceToHost);

    for(int i(0); i < length; ++i) {
        for(int j(0); j < length; ++j) {
            printf("%d ", CPUTable[(i * capacity) + j]);
        }
        printf("\n");
    }

    free(CPUTable);
}

// Magic if it works
__global__ void fillRow(char *table, char *sequence, int *index) {
    int column = threadIdx.x + blockIdx.x * blockDim.x;
    int row = *index;
    int pastSequencePos = row - (column + 1);
    int position = X(row, column);
    int position2 = (pastSequencePos > column) * X(pastSequencePos, column);

    table[position] += 1 + sequence[pastSequencePos] == sequence[row] * (((pastSequencePos > column) * (table[position2])) + (pastSequencePos <= column));
}

void initializeTable(char *table, char *sequence, int seqLength) {
    
    int *index;
    hipMalloc((void **)&index, sizeof(int));

    for(int i(0); i < seqLength; ++i) {
        hipMemcpy(index, (void *)&i, sizeof(int), hipMemcpyHostToDevice);
        fillRow<<< dim3(i + 1, 1), 1 >>>(table, sequence, index);
    }

    hipFree(index);
}

int main() {
    char *table;
    char *sequence;
    char *temps;
    bool *comparisons;
    int *size;
    int *cuda_capacity;
    int capacity = INITIAL_CAPACITY;

    // ((capacity + 1) * capacity) / 2 for table
    int table_size = (((capacity + 1) * capacity) / 2);

    // size needed for bool arrays in min and max functions
    int compare_size = (((capacity / 2) + 1) * (capacity / 2)) / 2;

    hipMalloc((void**)&table, table_size * sizeof(char));
    hipMalloc((void**)&sequence, capacity * sizeof(char));
    hipMalloc((void**)&temps, capacity * sizeof(char));
    hipMalloc((void**)&comparisons, compare_size * sizeof(bool));
    hipMalloc((void**)&size, sizeof(int));
    hipMalloc((void**)&cuda_capacity, sizeof(int));
    hipMemcpy(cuda_capacity, (int*)&capacity, sizeof(int), hipMemcpyHostToDevice);

    while (1) {

        hipMemset(table, 0, table_size * sizeof(int));
        hipMemset(comparisons, 0xFFFF, compare_size * sizeof(bool));
        
        char buffer[INITIAL_CAPACITY];
        printf("Input a sequence to curl:\n");
        scanf("%s", buffer);

        int seqLength = strlen(buffer);
        hipMemcpy(sequence, buffer, seqLength * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(size, (int*)&seqLength, sizeof(int), hipMemcpyHostToDevice);

        initializeTable(table, sequence, sequenceLength);

        clock_t start = clock();

        for(int i(0); i < capacity; i++) {
            findCurl(cudaSequence, table, seqLength, capacity);
            printf("curl = %d\n", curl);
            printTable(table, seqLength, capacity);
            sequence[seqLength] = curl;
            hipMemcpy(size, (int*)&seqLength, sizeof(int), hipMemcpyHostToDevice);
            sequenceByteSize = ++seqLength * sizeof(int);
            hipMalloc((void**)&cudaSequence, sequenceByteSize);
            hipMemcpy(cudaSequence, sequence, sequenceByteSize, hipMemcpyHostToDevice);
            fillRow<<< dim3(seqLength, 1), 1 >>>(cudaSequence, table, size, cap);
        }

        clock_t stop = clock();
        double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
        printf("Elapsed time: %.3fs\n", elapsed);
        printf("curl is %d\n\nsequence = ", curl);

        for(i = 0; i < seqLength; ++i){
            printf("%d ", sequence[i]);
        }

        printf("\n\n");
        hipFree(cudaSequence);
    }
    return 0;
}
